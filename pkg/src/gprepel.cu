#include "hip/hip_runtime.h"
//#####################################################################
//##  gpRepel : An R package for GPU computing
//##
//##  This program is free software; you can redistribute it and/or modify
//##  it under the terms of the GNU General Public License as published by
//##  the Free Software Foundation; version 3 of the License.
//##
//##  This program is distributed in the hope that it will be useful,
//##  but WITHOUT ANY WARRANTY; without even the implied warranty of
//##  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//##  GNU General Public License for more details.
//##
//##  You should have received a copy of the GNU General Public License
//##  along with this program; if not, write to the Free Software
//##  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA

#include <R.h>
#include "gpRepel.h"
#include "inc/gpRbase.h"
/*
#include <cusp/print.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
*/
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <thrust/sequence.h>
#include <thrust/gather.h>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <ostream>
#include <cstdlib>
#include <iostream>
#include <iomanip>

typedef thrust::tuple<Numeric,Numeric> 								Numeric2;
typedef typename thrust::device_vector<Numeric>::iterator         	NumericIterator;
typedef typename thrust::tuple<NumericIterator, NumericIterator>  	NumericIteratorTuple;
typedef typename thrust::zip_iterator<NumericIteratorTuple>       	Numeric2Iterator;
typedef thrust::tuple<Numeric,Numeric,Numeric> 						Numeric3;
typedef typename thrust::tuple<NumericIterator, NumericIterator, NumericIterator>  NumericIteratorTuple3;
typedef typename thrust::zip_iterator<NumericIteratorTuple3>       	Numeric3Iterator;

template <typename T>
struct is_less_than_zero
{
   __host__ __device__
   bool operator()(T x)
   {
      return x < 0;
   }
};

template <typename T>
struct is_greater_than
{
   T w;
   __host__ __device__
   is_greater_than(T w) : w(w) {}
   __host__ __device__
   bool operator()(T x)
   {
      return x > w;
   }
};

template <typename T>
struct plus_and_divide : public thrust::binary_function<T,T,T>
{
    T w;

    __host__ __device__
    plus_and_divide(T w) : w(w) {}

    __host__ __device__
    T operator()(const T& a, const T& b) const
    {
        return (a + b) / w;
    }
};

template <typename T>
struct minus_and_divide : public thrust::binary_function<T,T,T>
{
    T w;

    minus_and_divide(T w) : w(w) {}

    __host__ __device__
    T operator()(const T& a, const T& b) const
    {
        return (a - b) / w;
    }
};

template <typename T>
struct minus_and_divide_w : public thrust::binary_function<T,T,T>
{
    T w, maxb;

    __host__ __device__
    minus_and_divide_w(T w, T maxb) : w(w), maxb(maxb) {}

    __host__ __device__
    T operator()(const T& a, const T& b) const
    {
    	if(a < b)
        return (a - b) / w;
	else 
	return (a - maxb) / (w - a % maxb);
    }
};

// convert a linear index to a linear index in the transpose 
struct transpose_index : public thrust::unary_function<size_t,size_t>
{
    size_t m, n;

    __host__ __device__
    transpose_index(size_t _m, size_t _n) : m(_m), n(_n) {}

    __host__ __device__
    size_t operator()(size_t linear_index)
    {
        size_t j = linear_index / m;
        size_t i = linear_index % m;

        return j + i * n;
    }
};

// convert a linear index to a row index
struct column_index : public thrust::unary_function<size_t,size_t>
{
    size_t n;
    
    __host__ __device__
    column_index(size_t _n) : n(_n) {}

    __host__ __device__
    size_t operator()(size_t i)
    {
        return i / n;
    }
};

// convert a linear index to a row index
struct binary_index : public thrust::unary_function<size_t,size_t>
{
    size_t n;
    
    __host__ __device__
    binary_index(size_t _n) : n(_n) {}

    __host__ __device__
    size_t operator()(size_t i)
    {
        return (i / n) % 2;
    }
};

// transpose an M-by-N array
template <typename T>
void transpose(size_t m, size_t n, thrust::device_vector<T>& src, thrust::device_vector<T>& dst)
{
    thrust::counting_iterator<size_t> indices(0);
    
    thrust::gather(thrust::make_transform_iterator(indices, transpose_index(n, m)),
                   thrust::make_transform_iterator(indices, transpose_index(n, m)) + dst.size(),
                   src.begin(),
                   dst.begin());
}


// scan the rows of an M-by-N array
template <typename T>
void scan_horizontally(size_t m, size_t n, thrust::device_vector<T>& d_data)
{
    thrust::counting_iterator<size_t> indices(0);

    thrust::inclusive_scan_by_key(d_data.begin(), d_data.end(),
                                  thrust::make_transform_iterator(indices, column_index(m)),
                                  d_data.begin());
}

// print an M-by-N array
template <typename T>
void print(size_t m, size_t n, thrust::device_vector<T>& d_data)
{
    thrust::host_vector<T> h_data = d_data;

    for(size_t i = 0; i < m; i++)
    {
        for(size_t j = 0; j < n; j++)
            std::cout << std::setw(8) << h_data[i + j * m] << " ";
        std::cout << "\n";
    }
    std::cout << "\n";
}

// print an M-by-N array
template <typename T>
void printvec(size_t m, size_t n, thrust::device_vector<T>& d_data)
{
    thrust::host_vector<T> h_data = d_data;

    std::cout << "\n";
    for(size_t i = 0; i < m*n; i++)
        std::cout << std::setw(8) << h_data[i] << " ";
    std::cout << "\n";
    std::cout << "\n";
}

template <typename T>
void sumvec(thrust::device_vector<T>& gvec, Numeric& out)
{
    thrust::reduce(gvec.begin(), gvec.end(), out);
}

// This functor implements the dot product between 3d vectors
struct VecReorder : public thrust::binary_function<Numeric2,Numeric2,Numeric>
{
    Numeric w, maxb;
    Numeric ai,bi,av,bv, result;

    __host__ __device__
    VecReorder(Numeric w, Numeric maxb) : w(w), maxb(maxb) {}
    __host__ __device__
        Numeric operator()(const Numeric2& a, const Numeric2& b) const
        {
            Numeric ai=(int) thrust::get<0>(a) % (int) maxb;
            Numeric av=thrust::get<1>(a);
            Numeric bi=(int) thrust::get<0>(b) % (int) maxb;
            Numeric bv=thrust::get<1>(b);
	    int lastone = (int) maxb*((int) thrust::get<0>(a) / (int) maxb)-1;
	    if(ai > bi)
            return thrust::get<0>(a);
	    else
	    return lastone;
        }
};

// This functor implements the dot product between 3d vectors
struct  minus_and_divide_zip : public thrust::binary_function<Numeric3,Numeric3,Numeric>
{
    Numeric w, maxb;
    Numeric ai,bi,av,bv, result;

    __host__ __device__
    minus_and_divide_zip(Numeric w, Numeric maxb) : w(w), maxb(maxb) {}
    __host__ __device__
    Numeric operator()(const Numeric3& a, const Numeric3& b) const
    {
		Numeric ai=(int) thrust::get<0>(a) % (int) maxb;
		Numeric av=thrust::get<1>(a);
		Numeric bi=(int) thrust::get<0>(b) % (int) maxb;
		Numeric bv=thrust::get<1>(b);
	    int lastone = (int) maxb*((int) thrust::get<0>(a) / (int) maxb);
	    if(ai > bi)
                return (thrust::get<1>(a) - thrust::get<1>(b))/w;
	    else
	    	if((int)w - 1 != (int)thrust::get<0>(a) % (int)maxb)
	    		return (thrust::get<2>(b) - thrust::get<1>(b)) / ((int)w - (int)thrust::get<0>(a) % (int)maxb -1);
	    	else
	    		return 0;
	    		//return  ((thrust::get<2>(a)-thrust::get<1>(a))+(thrust::get<2>(b)-thrust::get<1>(b)))/w;
	    		//return  (thrust::get<2>(b)-thrust::get<1>(b))/w;
    }
};

template <typename InputVector, typename OutputVector>
void simple_moving_average(size_t m, size_t n, const InputVector& idata, size_t w, OutputVector& vout)
{
    typedef typename InputVector::value_type T;

    if (idata.size() < w)
        return;
    thrust::device_vector<size_t> output(m*n);
    thrust::device_vector<Numeric> voutput(m*n);
    thrust::device_vector<Numeric> data(m*n);
    thrust::device_vector<Numeric> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end());

    thrust::inclusive_scan(idata.begin(), idata.end(), data.begin());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin(), data.begin()));
    Numeric2Iterator firstw = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin() + w, data.begin() + w));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(vindex.end(),   data.end()));

    thrust::transform(firstw, last, first, output.begin(), VecReorder(w,m));
 
    thrust::gather(output.begin(), output.end(), data.begin(), voutput.begin());

    Numeric3Iterator first3 = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin(), data.begin(), voutput.begin()));
    Numeric3Iterator firstw3 = thrust::make_zip_iterator(thrust::make_tuple(vindex.begin() + w, data.begin() + w, voutput.begin() + w));
    Numeric3Iterator last3  = thrust::make_zip_iterator(thrust::make_tuple(vindex.end(), data.end(), voutput.end()));

    thrust::transform(firstw3, last3, first3, vout.begin(), minus_and_divide_zip(w,m));
    thrust::fill(vout.end()-w,vout.end(),vout[vout.size()-w-1]);
}


template <typename InputVector, typename OutputVector>
void double_moving_average(size_t m, size_t n, const InputVector& igva, size_t w, OutputVector& gvd)
{
    typedef typename InputVector::value_type T;
    if (igva.size() < w)
        return;

    thrust::device_vector<T> gva(igva.size());
    thrust::device_vector<T> gvb(igva.size());
    thrust::device_vector<T> gvc(igva.size());
    thrust::copy(igva.begin(), igva.end(), gva.begin());

    simple_moving_average(m,n,gva, w, gvb);
    thrust::reverse(gva.begin(), gva.end());
    simple_moving_average(m,n,gva, w, gvc);
    thrust::reverse(gvc.begin(), gvc.end());
    thrust::reverse(gva.begin(), gva.end());
    thrust::transform(gvc.begin(), gvc.end(), gvb.begin(), gvd.begin(), plus_and_divide<T>(T(2)));
}


//template <typename T>
void gprpostmave(PNumeric pint, PInteger a, PInteger b, PInteger win, PNumeric pout) {

    // window size of the moving average
    size_t w = win[0];
    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);

    simple_moving_average(m,n,gveca, w, gvecb);

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}

//template <typename T>
void gprpremave(PNumeric pint, PInteger a, PInteger b, PInteger win, PNumeric pout) {

    // window size of the moving average
    size_t w = win[0];
    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);

    thrust::reverse(gveca.begin(), gveca.end());
    simple_moving_average(m,n,gveca, w, gvecb);
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::reverse(gveca.begin(), gveca.end());

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}

//template <typename T>
void gprmoverage(PNumeric pint, PInteger a, PInteger b, PInteger win, PNumeric pout) {

    // window size of the moving average
    size_t w = win[0];
    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecz(a[0]*b[0]);
    double_moving_average(m,n,gveca,w,gvecz);

    // transfer data back to host
    thrust::copy(gvecz.begin(), gvecz.end(), pout);
}


//template <typename T>
void gprbasavoff(PNumeric pint, PInteger a, PInteger b, PInteger win1, PInteger win2, PNumeric pout) {

    // window size of the moving average
    size_t w1 = win1[0];//smaller window
    size_t w2 = win2[0];//larger window
    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecd(a[0]*b[0]);

    double_moving_average(m,n,gveca, w1, gvecb);
    double_moving_average(m,n,gveca, w2, gvecc);

    thrust::transform(gvecb.begin(), gvecb.end(), gvecc.begin(), gvecd.begin(), thrust::divides<Numeric>());

    // transfer data back to host
    thrust::copy(gvecd.begin(), gvecd.end(), pout);
}

//template <typename T>
void gprbasoroff(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {

    // window size of the moving average
    size_t w1 = win1[0];//smaller window

    size_t m = a[0];//row number
    size_t n = b[0];//column number

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);

    double_moving_average(m,n,gveca, w1, gvecb);

    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), gveca.begin(), thrust::divides<Numeric>());

    // transfer data back to host
    thrust::copy(gveca.begin(), gveca.end(), pout);
}

//template <typename T>
void gprdiff(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {
    size_t w = win1[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::transform(gveca.begin()+w, gveca.end(), gveca.begin(), gvecb.begin(), thrust::minus<Numeric>());

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}


//template <typename T>
void gprdiffrev(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {
    size_t w = win1[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::reverse(gveca.begin(), gveca.end());
    thrust::transform(gveca.begin()+w, gveca.end(), gveca.begin(), gvecb.begin(), thrust::minus<Numeric>());
    thrust::reverse(gvecb.begin(), gvecb.end());

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}


template <typename T>
struct oneup : public thrust::unary_function<T,T>
{
    T w;
	__host__ __device__
    oneup(T w) : w(w) {}

	__host__ __device__
    T operator()(const T& a) const
    {
    	if(a <= w)
    		return 0;
    	else
    		return a-w;
    }
};

template <typename T>
struct onedown : public thrust::unary_function<T,T>
{
    T w;
	__host__ __device__
    onedown(T w) : w(w) {}

    __host__ __device__
    T operator()(const T& a) const
    {
    	if(a >= w)
    		return 0;
    	else
    		return w-a;
    }
};


//template <typename T>
void gprup(PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric pout) {
	Numeric w1 = win1[0];

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), oneup<Numeric>(Numeric(w1)));

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}


//template <typename T>
void gprdown(PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric pout) {
	Numeric w1 = win1[0];

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), onedown<Numeric>(Numeric(w1)));

    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
}

template<typename T>
struct  minus_by: public thrust::unary_function<T,T>
{
    T w;
	__host__ __device__
    minus_by(T w) : w(w) {}

   __host__ __device__
   T operator()(const T &x) const
   {
    return (x - w)*(x - w);
   }
};

//template <typename T>
void gprsdall(PNumeric pint, PInteger a, PInteger b, PNumeric pout) {

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    Numeric sumall = thrust::reduce(gveca.begin(), gveca.end())/(a[0]*b[0]);
    Numeric result = thrust::transform_reduce(gveca.begin(), gveca.end(),
                                            minus_by<Numeric>(Numeric(sumall)),
                                            0,
                                            thrust::plus<Numeric>());
    pout[0] = sqrt(result/(a[0]*b[0]));
}

//template <typename T>
void gpravgall(PNumeric pint, PInteger a, PInteger b, PNumeric pout) {

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

	pout[0] = thrust::reduce(gveca.begin(), gveca.end())/(a[0]*b[0]);
}


// This functor implements the dot product between 3d vectors
struct kindcreate : public thrust::unary_function<Numeric,Numeric>
{
    Integer m, n;

    __host__ __device__
    kindcreate(Integer m, Integer n) : m(m), n(n) {}
    __host__ __device__
    Numeric operator()(const Numeric& a) const
    {
        //Integer ai=(int) ((int) a) % m;
        Integer av=(int) ((int) a) / m;
        //if(a > 0)
        return (int) av;
   }
};



//template <typename T>
void gprmovemax(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {

    // window size of the moving average
    //int w = win1[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());

    thrust::device_vector<Integer> vindex(a[0]*b[0]);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::transform(vindex.begin(), vindex.end(), gvecb.begin(), kindcreate(Integer(a[0]),Integer(b[0])));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>   binary_op;
    thrust::inclusive_scan_by_key(gvecb.begin(), gvecb.end(), gveca.begin(), gvecc.begin(),binary_pred,binary_op);
    thrust::reverse(gvecc.begin(), gvecc.end());
    thrust::inclusive_scan_by_key(gvecb.begin(), gvecb.end(), gvecc.begin(), gveca.begin(),binary_pred,binary_op);
    thrust::reverse(gveca.begin(), gveca.end());

    // transfer data back to host
    thrust::copy(gveca.begin(), gveca.end(), pout);
    //thrust::copy(vindex.begin(), vindex.end(), pout);
}

template <typename InputVector, typename OutputVector>
void meanorig(size_t m, size_t n, const InputVector& gveca, OutputVector& vout)
{
    thrust::device_vector<Numeric> orgbasoff(m*n);

    thrust::device_vector<Integer> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));
    thrust::device_vector<Numeric> gvecb(m*n);
//    thrust::device_vector<Numeric> gvecc(m*n);
    thrust::device_vector<Numeric> gvecd(m*n);

    double_moving_average(m,n,gveca, 150, gvecb);
    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), orgbasoff.begin(), thrust::divides<Numeric>());
    thrust::transform(orgbasoff.begin(), orgbasoff.end(), orgbasoff.begin(), oneup<Numeric>(Numeric(1)));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>  binary_max;
 /*
    thrust::device_vector<Numeric> avgbasoff(a[0]*b[0]);
    double_moving_average(a[0],b[0],gveca, 150, gvecb);
    double_moving_average(a[0],b[0],gveca,  80, gvecc);
    thrust::transform(gvecc.begin(), gvecc.end(), gvecb.begin(), avgbasoff.begin(), thrust::divides<Numeric>());
    thrust::transform(avgbasoff.begin(), avgbasoff.end(), avgbasoff.begin(), oneup<Numeric>(Numeric(1)));


    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), avgbasoff.begin(), gvecb.begin(),binary_pred,binary_max);
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), gvecb.begin(), gvecc.begin(),binary_pred,binary_max);
    thrust::reverse(gvecc.begin(), gvecc.end());
*/
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoff.begin(), gvecb.begin(),binary_pred,thrust::plus<Numeric>());
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), gvecb.begin(), gvecd.begin(),binary_pred,binary_max);
    thrust::reverse(gvecd.begin(), gvecd.end());
    thrust::fill(gvecb.begin(),gvecb.end(),m);
    thrust::transform(gvecd.begin(), gvecd.end(), gvecb.begin(), vout.begin(), thrust::divides<Numeric>());
}


void gprmeanmax(PNumeric pint, PInteger a, PInteger b, PInteger win1, PNumeric pout) {

    // window size of the moving average
    //int w = win1[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());
    thrust::device_vector<Numeric> meanvec(a[0]*b[0]);

    meanorig(a[0], b[0], gveca, meanvec);
    // transfer data back to host
    thrust::copy(meanvec.begin(), meanvec.end(), pout);
    //thrust::copy(vindex.begin(), vindex.end(), pout);
}


// This functor implements the dot product between 3d vectors
struct  compare_zip : public thrust::binary_function<Numeric2,Numeric2,Numeric>
{
     Numeric va,vb, da, db;

    __host__ __device__
    Numeric operator()(const Numeric2& a, const Numeric2& b) const
    {
		Numeric va=thrust::get<0>(a);
		Numeric da=thrust::get<1>(a);
		Numeric vb=thrust::get<0>(b);
		Numeric db=thrust::get<1>(b);
	    if(va > 0){
	    	if(da > 0 && db <0){
	    		return 1;
	    	}
	    	else
	    		return 0;
	    }
	    else
	    	return 0;
     }
};

// This functor implements division
struct  zipup : public thrust::unary_function<Numeric2,Numeric>
{
     Numeric va,da;

    __host__ __device__
    Numeric operator()(const Numeric2& a) const
    {
		Numeric va=thrust::get<0>(a);
		Numeric da=thrust::get<1>(a);
    	if(va <= da)
    		return 0;
    	else
    		return va-da;
     }
};


template <typename InputVector, typename OutputVector>
void find_maxpeaks(size_t m, size_t n, const InputVector& gveca, const Numeric& w1, const Numeric& w2, const Numeric& up, OutputVector& vout, OutputVector& avgbasoff, OutputVector& orgbasoff)
{
	//thrust::device_vector<Numeric> orgbasoff(m*n);

    thrust::device_vector<Integer> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));
    thrust::device_vector<Numeric> gvecb(m*n);
    thrust::device_vector<Numeric> gvecc(m*n);
    thrust::device_vector<Numeric> gvecd(m*n);

    double_moving_average(m,n,gveca, w2, gvecb);
    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), orgbasoff.begin(), thrust::divides<Numeric>());

    if(up > 0)
    	thrust::transform(orgbasoff.begin(), orgbasoff.end(), orgbasoff.begin(), oneup<Numeric>(Numeric(1)));
    else
    	thrust::transform(orgbasoff.begin(), orgbasoff.end(), orgbasoff.begin(), onedown<Numeric>(Numeric(1)));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>  binary_max;

    //thrust::device_vector<Numeric> avgbasoff(m*n);
    double_moving_average(m,n,gveca, w2, gvecb);
    double_moving_average(m,n,gveca, w1, gvecc);
    thrust::transform(gvecc.begin(), gvecc.end(), gvecb.begin(), avgbasoff.begin(), thrust::divides<Numeric>());
    if(up > 0)
    	thrust::transform(avgbasoff.begin(), avgbasoff.end(), avgbasoff.begin(), oneup<Numeric>(Numeric(1)));
    else
    	thrust::transform(avgbasoff.begin(), avgbasoff.end(), avgbasoff.begin(), onedown<Numeric>(Numeric(1)));

    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoff.begin(), gvecb.begin(),binary_pred,thrust::plus<Numeric>());
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), gvecb.begin(), gvecd.begin(),binary_pred,binary_max);
    thrust::reverse(gvecd.begin(), gvecd.end());
    thrust::fill(gvecb.begin(),gvecb.end(),m);
    thrust::transform(gvecd.begin(), gvecd.end(), gvecb.begin(), gvecd.begin(), thrust::divides<Numeric>());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoff.begin(), gvecd.begin()));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoff.end(),   gvecd.end()));

    thrust::transform(first, last, gvecb.begin(), zipup());

    thrust::transform(gvecb.begin()+1, gvecb.end(), gvecb.begin(), gvecc.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,gvecc, 20, gvecd);

    Numeric2Iterator first0 = thrust::make_zip_iterator(thrust::make_tuple(gvecb.begin(), gvecd.begin()));
    Numeric2Iterator first1 = thrust::make_zip_iterator(thrust::make_tuple(gvecb.begin() + 1, gvecd.begin() + 1));
    Numeric2Iterator last0  = thrust::make_zip_iterator(thrust::make_tuple(gvecb.end(),  gvecd.end()));

    thrust::transform(first0, last0, first1, vout.begin(), compare_zip());

}

//template <typename T>
void gprpeakmask(PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric win2, PNumeric up, PNumeric pout) {
    size_t m = a[0];//row number
    size_t n = b[0];//column number
    int w1 = win1[0];//difference window
    int w2 = win2[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());
    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecd(a[0]*b[0]);

    find_maxpeaks(m,n,gveca,w1,w2,up[0],gvecb,gvecc,gvecd);
    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
    thrust::copy(gvecc.begin(), gvecc.end(), pout+m*n+1);
    thrust::copy(gvecd.begin(), gvecd.end(), pout+2*m*n+1);
}

// This functor implements mask
struct  upmask : public thrust::unary_function<Numeric,Numeric>
{
	Numeric out;
	__host__ __device__
    upmask(Numeric out) : out(out) {}
    __host__ __device__
    Numeric operator()(const Numeric& a)
    {
    	if(a > 0){
    		return out;
    	}
    	else
    	{
    		out=out+1;
    		return 0;
    	}
     }
};


template <typename InputVector, typename OutputVector>
void find_doublemaxpeaks(size_t m, size_t n, const InputVector& gveca, const Numeric& w1, const Numeric& w2, const Numeric& w3, OutputVector& pout, OutputVector& avgbasoffp, OutputVector& dout, OutputVector& avgbasoffd)
{
	thrust::device_vector<Numeric> orgbasoffp(m*n);
	thrust::device_vector<Numeric> orgbasoffd(m*n);

    thrust::device_vector<Numeric> gvecb(m*n);
    thrust::device_vector<Numeric> gvecc(m*n);

    double_moving_average(m,n,gveca, w2, gvecb);
    thrust::transform(gveca.begin(), gveca.end(), gvecb.begin(), gvecc.begin(), thrust::divides<Numeric>());

    thrust::transform(gvecc.begin(), gvecc.end(), orgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(gvecc.begin(), gvecc.end(), orgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

    thrust::equal_to<Numeric> binary_pred;
    thrust::maximum<Numeric>  binary_max;

    //thrust::device_vector<Numeric> avgbasoffp(m*n);
    //thrust::device_vector<Numeric> avgbasoffd(m*n);
    thrust::device_vector<Numeric> gvecd(m*n);
    double_moving_average(m,n,gveca, w2, gvecb);
    double_moving_average(m,n,gveca, w1, gvecc);
    thrust::transform(gvecc.begin(), gvecc.end(), gvecb.begin(), gvecd.begin(), thrust::divides<Numeric>());

    thrust::transform(gvecd.begin(), gvecd.end(), avgbasoffp.begin(), oneup<Numeric>(Numeric(1)));
    thrust::transform(gvecd.begin(), gvecd.end(), avgbasoffd.begin(), onedown<Numeric>(Numeric(1)));

    thrust::device_vector<Integer> vindex(m*n);
    thrust::sequence(vindex.begin(),vindex.end(),0);
    thrust::transform(vindex.begin(), vindex.end(), vindex.begin(), kindcreate(Integer(m),Integer(n)));


    //orgbasoffp, avgbasoffp
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffp.begin(), gvecb.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), gvecb.begin(), gvecd.begin(),binary_pred,binary_max);
    thrust::reverse(gvecd.begin(), gvecd.end());
    thrust::fill(gvecb.begin(),gvecb.end(),m/(80/w1));
    thrust::transform(gvecd.begin(), gvecd.end(), gvecb.begin(), gvecd.begin(), thrust::divides<Numeric>());

    Numeric2Iterator first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.begin(), gvecd.begin()));
    Numeric2Iterator last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffp.end(),   gvecd.end()));

    thrust::transform(first, last, gvecb.begin(), zipup());

    thrust::transform(gvecb.begin()+1, gvecb.end(), gvecb.begin(), gvecc.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,gvecc, w3, gvecd);

    Numeric2Iterator first0 = thrust::make_zip_iterator(thrust::make_tuple(gvecb.begin(), gvecd.begin()));
    Numeric2Iterator first1 = thrust::make_zip_iterator(thrust::make_tuple(gvecb.begin() + 1, gvecd.begin() + 1));
    Numeric2Iterator last0  = thrust::make_zip_iterator(thrust::make_tuple(gvecb.end(),  gvecd.end()));

    thrust::transform(first0, last0, first1, pout.begin(), compare_zip());

    //orgbasoffd, avgbasoffd
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), orgbasoffd.begin(), gvecb.begin(), binary_pred,thrust::plus<Numeric>());
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::inclusive_scan_by_key(vindex.begin(), vindex.end(), gvecb.begin(), gvecd.begin(),binary_pred,binary_max);
    thrust::reverse(gvecd.begin(), gvecd.end());
    thrust::fill(gvecb.begin(),gvecb.end(),m/(80/w1));
    thrust::transform(gvecd.begin(), gvecd.end(), gvecb.begin(), gvecd.begin(), thrust::divides<Numeric>());

    first = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.begin(), gvecd.begin()));
    last  = thrust::make_zip_iterator(thrust::make_tuple(avgbasoffd.end(),   gvecd.end()));

    thrust::transform(first, last, gvecb.begin(), zipup());

    thrust::transform(gvecb.begin()+1, gvecb.end(), gvecb.begin(), gvecc.begin(), thrust::minus<Numeric>());
    double_moving_average(m,n,gvecc, w3, gvecd);

    first0 = thrust::make_zip_iterator(thrust::make_tuple(gvecb.begin(), gvecd.begin()));
    first1 = thrust::make_zip_iterator(thrust::make_tuple(gvecb.begin() + 1, gvecd.begin() + 1));
    last0  = thrust::make_zip_iterator(thrust::make_tuple(gvecb.end(),  gvecd.end()));

    thrust::transform(first0, last0, first1, dout.begin(), compare_zip());
    /*
    thrust::transform(avgbasoffd.begin(), avgbasoffd.end(), avgbasoffd.begin(), upmask(Numeric(1)));
    thrust::inclusive_scan_by_key(gvecb.begin(), gvecb.end(), dout.begin(), gvecc.begin(),binary_pred,binary_max);
    thrust::reverse(gvecb.begin(), gvecb.end());
    thrust::reverse(gvecc.begin(), gvecc.end());
    thrust::inclusive_scan_by_key(gvecb.begin(), gvecb.end(), gvecc.begin(), avgbasoffd.begin(),binary_pred,binary_max);
    thrust::reverse(avgbasoffd.begin(), avgbasoffd.end());
    */
}

//template <typename T>
void gprpeak2mask(PNumeric pint, PInteger a, PInteger b, PNumeric win1, PNumeric win2, PNumeric win3, PNumeric pout) {
    size_t m = a[0];//row number
    size_t n = b[0];//column number
    int w1 = win1[0];//window
    int w2 = win2[0];//baseline window
    int w3 = win3[0];//difference window

    // transfer data to the device
    thrust::device_vector<Numeric> gveca(a[0]*b[0]);
    thrust::copy(pint,pint+a[0]*b[0],gveca.begin());
    thrust::device_vector<Numeric> gvecb(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecc(a[0]*b[0]);
    thrust::device_vector<Numeric> gvecd(a[0]*b[0]);
    thrust::device_vector<Numeric> gvece(a[0]*b[0]);

    find_doublemaxpeaks(m,n,gveca,w1,w2,w3,gvecb,gvecc,gvecd,gvece);
    // transfer data back to host
    thrust::copy(gvecb.begin(), gvecb.end(), pout);
    thrust::copy(gvecc.begin(), gvecc.end(), pout+m*n+1);
    thrust::copy(gvecd.begin(), gvecd.end(), pout+2*m*n+1);
    thrust::copy(gvece.begin(), gvece.end(), pout+3*m*n+1);
}

